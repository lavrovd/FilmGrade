
#include <hip/hip_runtime.h>
__global__ void FilmGradeKernel(int p_Width, int p_Height, float p_ExpR, float p_ExpG, float p_ExpB, 
    float p_ContR, float p_ContG, float p_ContB, float p_SatR, float p_SatG, float p_SatB, 
    float p_ShadR, float p_ShadG, float p_ShadB, float p_MidR, float p_MidG, float p_MidB, 
    float p_HighR, float p_HighG, float p_HighB, float p_ShadP, float p_HighP, float p_ContP, 
    float p_Display, const float* p_Input, float* p_Output)
{
   const int x = blockIdx.x * blockDim.x + threadIdx.x;
   const int y = blockIdx.y * blockDim.y + threadIdx.y;

   if ((x < p_Width) && (y < p_Height))
   {
       const int index = ((y * p_Width) + x) * 4;
       
       float e = 2.718281828459045;
       float pie = 3.141592653589793;
       
	   float width = p_Width;
       float height = p_Height;    	
            	  
	   float Red = p_Display != 1.0f ? p_Input[index + 0] : x / width;
	   float Green = p_Display != 1.0f ? p_Input[index + 1] : x / width;
	   float Blue = p_Display != 1.0f ? p_Input[index + 2] : x / width;
	   
	   float expR = Red + p_ExpR/100;
	   float expG = Green + p_ExpG/100;
	   float expB = Blue + p_ExpB/100;
	   
	   float contR = (expR - p_ContP) * p_ContR + p_ContP;
	   float contG = (expG - p_ContP) * p_ContG + p_ContP;
	   float contB = (expB - p_ContP) * p_ContB + p_ContP;
	   
	   float luma = contR * 0.2126f + contG * 0.7152f + contB * 0.0722f;
	   float satR = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * luma + contR * p_SatR;
	   float satG = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * luma + contG * p_SatG;
	   float satB = (1.0f - (p_SatR*0.2126f + p_SatG* 0.7152f + p_SatB*0.0722f)) * luma + contB * p_SatB;
	   
	   float expr1 = (p_ShadP / 2.0f) - (1.0f - p_HighP)/4.0f;
	   float expr2 = (1.0f - (1.0f - p_HighP)/2.0f) + (p_ShadP / 4.0f);
	   float expr3R = (satR - expr1) / (expr2 - expr1);
	   float expr3G = (satG - expr1) / (expr2 - expr1);
	   float expr3B = (satB - expr1) / (expr2 - expr1);
	   float expr4 =  p_ContP < 0.5f ? 0.5f - (0.5f - p_ContP)/2.0f : 0.5f + (p_ContP - 0.5f)/2.0f;
	   float expr5R = expr3R > expr4 ? (expr3R - expr4) / (2.0f - 2.0f*expr4) + 0.5f : expr3R /(2.0f*expr4);
	   float expr5G = expr3G > expr4 ? (expr3G - expr4) / (2.0f - 2.0f*expr4) + 0.5f : expr3G /(2.0f*expr4);
	   float expr5B = expr3B > expr4 ? (expr3B - expr4) / (2.0f - 2.0f*expr4) + 0.5f : expr3B /(2.0f*expr4);
	   float expr6R = (((sin(2.0f * pie * (expr5R -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidR*4) + expr3R;
	   float expr6G = (((sin(2.0f * pie * (expr5G -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidG*4) + expr3G;
	   float expr6B = (((sin(2.0f * pie * (expr5B -1.0f/4.0f)) + 1.0f) / 20.0f) * p_MidB*4) + expr3B;
	   float midR = satR >= expr1 && satR <= expr2 ? expr6R * (expr2 - expr1) + expr1 : satR;
	   float midG = satG >= expr1 && satG <= expr2 ? expr6G * (expr2 - expr1) + expr1 : satG;
	   float midB = satB >= expr1 && satB <= expr2 ? expr6B * (expr2 - expr1) + expr1 : satB;

	   float shadupR1 = 2.0f * (midR/p_ShadP) - log((midR/p_ShadP) * (e * p_ShadR * 2.0f) + 1.0f)/log(e * p_ShadR * 2.0f + 1.0f);
	   float shadupR = midR <= p_ShadP && p_ShadR > 0.0f ? (shadupR1 + p_ShadR * (1.0f - shadupR1)) * p_ShadP : midR;
	   float shadupG1 = 2.0f * (midG/p_ShadP) - log((midG/p_ShadP) * (e * p_ShadG * 2.0f) + 1.0f)/log(e * p_ShadG * 2.0f + 1.0f);
	   float shadupG = midG <= p_ShadP && p_ShadG > 0.0f ? (shadupG1 + p_ShadG * (1.0f - shadupG1)) * p_ShadP : midG;
	   float shadupB1 = 2.0f * (midB/p_ShadP) - log((midB/p_ShadP) * (e * p_ShadB * 2.0f) + 1.0f)/log(e * p_ShadB * 2.0f + 1.0f);
	   float shadupB = midB <= p_ShadP && p_ShadB > 0.0f ? (shadupB1 + p_ShadB * (1.0f - shadupB1)) * p_ShadP : midB;
	   
	   float shaddownR1 = shadupR/p_ShadP + p_ShadR*2 * (1.0f - shadupR/p_ShadP);
	   float shaddownR = shadupR <= p_ShadP && p_ShadR < 0.0f ? (log(shaddownR1 * (e * p_ShadR * -2.0f) + 1.0f)/log(e * p_ShadR * -2.0f + 1.0f)) * p_ShadP : shadupR;
	   float shaddownG1 = shadupG/p_ShadP + p_ShadG*2 * (1.0f - shadupG/p_ShadP);
	   float shaddownG = shadupG <= p_ShadP && p_ShadG < 0.0f ? (log(shaddownG1 * (e * p_ShadG * -2.0f) + 1.0f)/log(e * p_ShadG * -2.0f + 1.0f)) * p_ShadP : shadupG;
	   float shaddownB1 = shadupB/p_ShadP + p_ShadB*2 * (1.0f - shadupB/p_ShadP);
	   float shaddownB = shadupB <= p_ShadP && p_ShadB < 0.0f ? (log(shaddownB1 * (e * p_ShadB * -2.0f) + 1.0f)/log(e * p_ShadB * -2.0f + 1.0f)) * p_ShadP : shadupB;
	   
	   float highupR1 = ((shaddownR - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighR * 2.0f));
	   float highupR = shaddownR >= p_HighP && p_HighR > 0.0f ? (2.0f * highupR1 - log(highupR1 * e * p_HighR + 1.0f)/log(e * p_HighR + 1.0f)) * (1.0f - p_HighP) + p_HighP : shaddownR;
	   float highupG1 = ((shaddownG - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighG * 2.0f));
	   float highupG = shaddownG >= p_HighP && p_HighG > 0.0f ? (2.0f * highupG1 - log(highupG1 * e * p_HighG + 1.0f)/log(e * p_HighG + 1.0f)) * (1.0f - p_HighP) + p_HighP : shaddownG;
	   float highupB1 = ((shaddownB - p_HighP) / (1.0f - p_HighP)) * (1.0f + (p_HighB * 2.0f));
	   float highupB = shaddownB >= p_HighP && p_HighB > 0.0f ? (2.0f * highupB1 - log(highupB1 * e * p_HighB + 1.0f)/log(e * p_HighB + 1.0f)) * (1.0f - p_HighP) + p_HighP : shaddownB;
	   
	   float highdownR1 = (highupR - p_HighP) / (1.0f - p_HighP);
	   float highdownR = highupR >= p_HighP && p_HighR < 0.0f ? log(highdownR1 * (e * p_HighR * -2.0f) + 1.0f)/log(e * p_HighR * -2.0f + 1.0f) * (1.0f + p_HighR) * (1.0f - p_HighP) + p_HighP : highupR;
	   float highdownG1 = (highupG - p_HighP) / (1.0f - p_HighP);
	   float highdownG = highupG >= p_HighP && p_HighG < 0.0f ? log(highdownG1 * (e * p_HighG * -2.0f) + 1.0f)/log(e * p_HighG * -2.0f + 1.0f) * (1.0f + p_HighG) * (1.0f - p_HighP) + p_HighP : highupG;
	   float highdownB1 = (highupB - p_HighP) / (1.0f - p_HighP);
	   float highdownB = highupB >= p_HighP && p_HighB < 0.0f ? log(highdownB1 * (e * p_HighB * -2.0f) + 1.0f)/log(e * p_HighB * -2.0f + 1.0f) * (1.0f + p_HighB) * (1.0f - p_HighP) + p_HighP : highupB;
	   
	   float outR = p_Display != 1.0f ? highdownR : y / height >= p_ShadP && y / height <= p_ShadP + 0.005f ? (fmodf(x, 2.0f) != 0.0f ? 1.0f : 0) : highdownR >= (y - 5) / height && highdownR <= (y + 5) / height ? 1.0f : 0.0f;
	   float outG = p_Display != 1.0f ? highdownG : y / height >= p_HighP && y / height <= p_HighP + 0.005f ? (fmodf(x, 2.0f) != 0.0f ? 1.0f : 0) : highdownG >= (y - 5) / height && highdownG <= (y + 5) / height ? 1.0f : 0.0f;
	   float outB = p_Display != 1.0f ? highdownB : y / height >= p_ContP && y / height <= p_ContP + 0.005f ? (fmodf(x, 2.0f) != 0.0f ? 1.0f : 0) : highdownB >= (y - 5) / height && highdownB <= (y + 5) / height ? 1.0f : 0.0f;
				 			
       p_Output[index + 0] = outR;
       p_Output[index + 1] = outG;
       p_Output[index + 2] = outB;
       p_Output[index + 3] = p_Input[index + 3];
   }
}

void RunCudaKernel(int p_Width, int p_Height, float* p_Exp, float* p_Cont, float* p_Sat, 
float* p_Shad, float* p_Mid, float* p_High, float* p_Pivot, float* p_Display, const float* p_Input, float* p_Output)
{
    dim3 threads(128, 1, 1);
    dim3 blocks(((p_Width + threads.x - 1) / threads.x), p_Height, 1);

    FilmGradeKernel<<<blocks, threads>>>(p_Width, p_Height, p_Exp[0], p_Exp[1], p_Exp[2], p_Cont[0], p_Cont[1], p_Cont[2], 
    p_Sat[0], p_Sat[1], p_Sat[2], p_Shad[0], p_Shad[1], p_Shad[2], p_Mid[0], p_Mid[1], p_Mid[2], p_High[0], p_High[1], p_High[2], 
    p_Pivot[0], p_Pivot[1], p_Pivot[2], p_Display[0], p_Input, p_Output);
}
